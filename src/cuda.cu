#include "hip/hip_runtime.h"
/*******************************************************************************
File: seq.c
Created by: CJ Dimaano
Date created: March 29, 2017
*******************************************************************************/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

extern "C" {
#include "data.h"
#include "mem.h"
}

/** Declarations **************************************************************/

static int train(
    double * const x,
    double * const y,
    const int count,
    const int epochs,
    const double c,
    const double gamma0,
    const double s,
    double * const w
);
static void test(
    const double * const x,
    const double * const y,
    const int count,
    const double * const w
);
static double getPrediction(
    const double * const x_i,
    const double * const w
);
static int parseArgs(const int, char **, int *, double *, double *, double *);

/** Main **********************************************************************/

int main(int argc, char **argv) {
    double *w, *x, *y, c = 1 << 10, gamma0 = 0.01, s = 10e5;
    int ret, epochs = 200;

    /*** Parse command-line arguments. ***/
    ret = parseArgs(argc, argv, &epochs, &c, &gamma0, &s);
    if(ret < 0) {
        return -1;
    }
    printf("epochs: %d\n", epochs);
    printf("C: %f\n", c);
    printf("gamma0: %f\n", gamma0);
    printf("s: %f\n", s);

    /*** Allocate memory. ***/
    if(init(&x, &y, &w) < 0) {
        return -2;
    }

    /*** Load training data. ***/
    ret = load(TRAIN_SET, x, y);
    if(ret < 0) {
        cleanup(&x, &y, &w);
        return -3;
    }

    /*** Train classifier. ***/
    ret = train(x, y, ret, epochs, c, gamma0, s, w);
    if(ret < 0) {
        cleanup(&x, &y, &w);
        return -4;
    }

    /*** Load test data. ***/
    ret = load(TEST_SET, x, y);
    if(ret < 0) {
        cleanup(&x, &y, &w);
        return ret;
    }

    /*** Test classifier accuracy. ***/
    test(x, y, ret, w);

    /*** Cleanup memory from examples. ***/
    cleanup(&x, &y, &w);

    return 0;
}

/** Static functions **********************************************************/

extern __global__ void trainCompute(double *p_w, double *p_x, double gamma0, int feature_count, double a, double b, double c, double e, int i, double t) {
  int bx = blockIdx.x;
  int tx = threadIdx.x;

  p_w[tx + 32 * bx] = p_w[tx + 32 * bx] - (gamma0 / (1 + gamma0 * t / c)) * (a * p_x[i * feature_count + tx + 32 * bx] + b * p_w[tx + 32 * bx]);

  __syncthreads();
}

/**
 * train
 *
 * @summary
 *   Trains the weights of the classifier.
 */
static int train(
    double * const x,
    double * const y,
    const int count,
    const int epochs,
    const double c,
    const double gamma0,
    const double s,
    double * const w
) {
    int epoch, i, j;
    double a, b = 2 / (s * s), t = 1, dot, e;

    // fillWeights(w);

    for(epoch = 0; epoch < epochs; epoch++) {
        shuffle(count, x, y);
        for(i = 0; i < count; i++) {
            dot = 0;
            for(j = 0; j < FEATURE_COUNT; j++)
                dot += x[i * FEATURE_COUNT + j] * w[j];
            e = exp(-y[i] * dot);
            a = -y[i] * e / (1 + e);

            // Allocate buffers on the GPU.
            double * p_x;
            double * p_w;

            // TODO: Probably want to make these constants in the mem.h file instead of re-computing them.
            int x_size = MAX_EXAMPLES * FEATURE_COUNT * sizeof(double);
            int w_size = FEATURE_COUNT * sizeof(double);

            hipMalloc((void **)&p_x, x_size);
            hipMalloc((void **)&p_w, w_size);

            // Copy host buffers into device buffers.
            hipMemcpy(p_x, x, x_size, hipMemcpyHostToDevice);
            hipMemcpy(p_w, w, w_size, hipMemcpyHostToDevice);

            // Perform the computation.
            dim3 dimGrid((FEATURE_COUNT+31)/32, 1);
            dim3 dimBlock(32, 1);

            trainCompute<<<dimGrid,dimBlock>>>(p_w, p_x, gamma0, FEATURE_COUNT, a, b, c, e, i, t);

            // Copy the result off of the GPU.
            hipMemcpy(w, p_w, w_size, hipMemcpyDeviceToHost);

            // Free the created buffers.
            hipFree(p_x);
            hipFree(p_w);

            t += 1.0;
        }
    }

    return 0;
}

/**
 * test
 */
static void test(
    const double * const x,
    const double * const y,
    const int count,
    const double * const w
) {
    int i;
    const double *x_i = x;
    double y_i, y_p, p, r, f1, accuracy;
    /*** True/False Positive/Negative ***/
    int tp = 0;
    int fp = 0;
    int tn = 0;
    int fn = 0;

    for(i = 0; i < count; i++) {
        y_i = y[i];
        y_p = getPrediction(x_i, w);
        if(y_i > 0 && y_p > 0)
            tp++;
        else if(y_i > 0 && y_p < 0)
            fn++;
        else if(y_i < 0 && y_p > 0)
            fp++;
        else
            tn++;
        x_i = (x_i + FEATURE_COUNT);
    }

    p = 0;
    r = 0;
    f1 = 0;
    if(tp > 0) {
        p = (double)tp / (double)(tp + fp);
        r = (double)tp / (double)(tp + fn);
        f1 = 2 * p * r / (p + r);
    }
    else {
        if(fp == 0)
            p = 1;
        if(fn == 0)
            r = 1;
    }

    accuracy = (double)(tp + tn) / (double)count;
    printf("tp, fp, tn, fn: %d, %d, %d, %d\n", tp, fp, tn, fn);
    printf("accuracy: %f\nf1: %f\n", accuracy, f1);
}

/**
 * getPrediction
 */
static double getPrediction(
    const double * const x_i,
    const double * const w
) {
    int i;
    double dot = 0;
    for(i = 0; i < FEATURE_COUNT; i++)
        dot += x_i[i] * w[i];
    return dot > 0 ? 1 : -1;
}

/**
 * printUsage
 */
static void printUsage(const char *prgm) {
    printf("usage:\n");
    printf("\t%s [-e <int>] [-C <double>] [-s <double>] [-g <double>]\n\n", prgm);
    printf("Options:\n");
    printf("\t-e <int>       Specifies the number of epochs over which to"
        " train.\n");
    printf("\t-C <double>    Specifies the C hyper parameter.\n");
    printf("\t-g <double>    Specifies the gamma0 hyper parameter.\n");
    printf("\t-s <double>    Specifies the s hyper parameter.\n");
}

/**
 * parseArgs
 */
static int parseArgs(
    const int argc,
    char **argv,
    int *epochs,
    double *c,
    double *gamma0,
    double *s
) {
    int i;
    for(i = 1; i < argc; i++) {
        /*** epochs ***/
        if(strcmp(argv[i], "-e") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -1;
            }
            (*epochs) = atoi(argv[i]);
            if((*epochs) < 1) {
                fprintf(stderr, "error: number of epochs must be greater than"
                    " 0\n");
                printUsage(argv[0]);
                return -2;
            }
        }
        /*** C ***/
        else if(strcmp(argv[i], "-C") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -3;
            }
            (*c) = atof(argv[i]);
            if((*c) <= 0) {
                fprintf(stderr, "error: C must be positive\n");
                printUsage(argv[0]);
                return -4;
            }
        }
        /*** gamma0 ***/
        else if(strcmp(argv[i], "-g") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -5;
            }
            (*gamma0) = atof(argv[i]);
            if((*gamma0) <= 0) {
                fprintf(stderr, "error: gamma0 must be positive\n");
                printUsage(argv[0]);
                return -6;
            }
        }
        /*** s ***/
        else if(strcmp(argv[i], "-s") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -7;
            }
            (*s) = atof(argv[i]);
            if((*s) <= 0) {
                fprintf(stderr, "error: s must be positive\n");
                printUsage(argv[0]);
                return -8;
            }
        }
        /*** unexpected argument ***/
        else {
            fprintf(stderr, "error: unexpected argument\n");
            printUsage(argv[0]);
            return -9;
        }
    }
    return 0;
}
