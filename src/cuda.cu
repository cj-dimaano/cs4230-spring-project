#include "hip/hip_runtime.h"
/*******************************************************************************
File: seq.c
Created by: CJ Dimaano
Date created: March 29, 2017
*******************************************************************************/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

extern "C" {
#include "data.h"
#include "mem.h"
}

/** Declarations **************************************************************/

static int train(
    double * const x,
    double * const y,
    const int count,
    const int epochs,
    const double c,
    const double gamma0,
    const double s,
    double * const w
);
static void test(
    const double * const x,
    const double * const y,
    const int count,
    const double * const w
);
static double getPrediction(
    const double * const x_i,
    const double * const w
);
static int parseArgs(const int, char **, int *, double *, double *, double *);

/** Main **********************************************************************/

int main(int argc, char **argv) {
    double *w, *x, *y, c = 1 << 10, gamma0 = 0.01, s = 10e5;
    int ret, epochs = 200;

    /*** Parse command-line arguments. ***/
    ret = parseArgs(argc, argv, &epochs, &c, &gamma0, &s);
    if(ret < 0) {
        return -1;
    }
    printf("epochs: %d\n", epochs);
    printf("C: %f\n", c);
    printf("gamma0: %f\n", gamma0);
    printf("s: %f\n", s);

    /*** Allocate memory. ***/
    if(init(&x, &y, &w) < 0) {
        return -2;
    }

    /*** Load training data. ***/
    ret = load(TRAIN_SET, x, y);
    if(ret < 0) {
        cleanup(&x, &y, &w);
        return -3;
    }

    /*** Train classifier. ***/
    ret = train(x, y, ret, epochs, c, gamma0, s, w);
    if(ret < 0) {
        cleanup(&x, &y, &w);
        return -4;
    }

    /*** Load test data. ***/
    ret = load(TEST_SET, x, y);
    if(ret < 0) {
        cleanup(&x, &y, &w);
        return ret;
    }

    /*** Test classifier accuracy. ***/
    test(x, y, ret, w);

    /*** Cleanup memory from examples. ***/
    cleanup(&x, &y, &w);

    return 0;
}

/** Static functions **********************************************************/

extern __global__ void trainCompute(double *p_w, double *p_x, double gamma0, double a, double b, double c, double e, int i, double t) {
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int j = tx + 32 * bx;

  if (j < FEATURE_COUNT) {
    p_w[j] = p_w[j] - (gamma0 / (1 + gamma0 * t / c)) * (a * p_x[i * FEATURE_COUNT + j] + b * p_w[j]);
  }

  __syncthreads();
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

int compare(double *a, double *b, int size, double threshold) {
  int i;
  for (i=0; i<size; i++) {
    if (abs(a[i]-b[i]) > threshold) return 0;
  }
  return 1;
}


/**
 * train
 *
 * @summary
 *   Trains the weights of the classifier.
 */
static int train(
    double * const x,
    double * const y,
    const int count,
    const int epochs,
    const double c,
    const double gamma0,
    const double s,
    double * const w
) {
    int epoch, i, j;
    double a, b = 2 / (s * s), t = 1, dot, e;

    // fillWeights(w);

    // Allocate buffers on the GPU.
    double * p_x;
    double * p_w;

    // TODO: Probably want to make these constants in the mem.h file instead of re-computing them.
    int x_size = MAX_EXAMPLES * FEATURE_COUNT * sizeof(double);
    int w_size = FEATURE_COUNT * sizeof(double);

    hipMalloc((void **)&p_x, x_size);
    hipMalloc((void **)&p_w, w_size);

    // Copy host buffers into device buffers.
    hipMemcpy(p_x, x, x_size, hipMemcpyHostToDevice);
    hipMemcpy(p_w, w, w_size, hipMemcpyHostToDevice);

    // double * temp_x = (double *) malloc(x_size);
    // hipMemcpy(temp_x, p_x, x_size, hipMemcpyDeviceToHost);
    // int res = compare(temp_x, x, MAX_EXAMPLES * FEATURE_COUNT, 0.001);
    // if (res == 0) {
    //   printf("Mismatch between X.\n");
    //   printf("Index Sequential Parallel\n");
    //   for (j = 0; j < MAX_EXAMPLES * FEATURE_COUNT; j++) {
    //     printf("%d %f %f\n", j, x[j], temp_x[j]);
    //   }
    // } {
    //   printf("No mismatch between x arrays.\n");
    // }

    // free(temp_x);

    // Temporary buffer for verifying the output of the GPU.
    double * temp_w = (double *) malloc(w_size);

    // Perform the computation.
    dim3 dimGrid((FEATURE_COUNT+31)/32, 1);
    dim3 dimBlock(32, 1);

    for(epoch = 0; epoch < epochs; epoch++) {
        shuffle(count, x, y);
        hipMemcpy(p_x, x, x_size, hipMemcpyHostToDevice);
        for(i = 0; i < count; i++) {
            dot = 0;
            for(j = 0; j < FEATURE_COUNT; j++)
                dot += x[i * FEATURE_COUNT + j] * w[j];
            e = exp(-y[i] * dot);
            a = -y[i] * e / (1 + e);

            trainCompute<<<dimGrid,dimBlock>>>(p_w, p_x, gamma0, a, b, c, e, i, t);
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );

            // for(j = 0; j < FEATURE_COUNT; j++) {
            //   w[j] = w[j] - (gamma0 / (1 + gamma0 * t / c)) * (a * x[i * FEATURE_COUNT + j] + b * w[j]);
            // }


            // Copy p_w to the temporary buffer and compare it with the result of the sequential version.
            // hipMemcpy(temp_w, p_w, w_size, hipMemcpyDeviceToHost);
            // int res = compare(temp_w, w, FEATURE_COUNT, 0.001);
            // if (res == 0) {
            //   printf("Mismatch between results.\n");
            //   printf("Index Sequential Parallel\n");
            //   for (j = 0; j < FEATURE_COUNT; j++) {
            //     printf("%d %f %f\n", j, w[j], temp_w[j]);
            //   }
            //   exit(1);
            // }
            // exit(1);

            hipMemcpy(w, p_w, w_size, hipMemcpyDeviceToHost);

            t += 1.0;
        }

        // free(temp_w);
    }

    // Copy the result off of the GPU.
    hipMemcpy(w, p_w, w_size, hipMemcpyDeviceToHost);

    // Free the created buffers.
    hipFree(p_x);
    hipFree(p_w);

    return 0;
}

/**
 * test
 */
static void test(
    const double * const x,
    const double * const y,
    const int count,
    const double * const w
) {
    int i;
    const double *x_i = x;
    double y_i, y_p, p, r, f1, accuracy;
    /*** True/False Positive/Negative ***/
    int tp = 0;
    int fp = 0;
    int tn = 0;
    int fn = 0;

    for(i = 0; i < count; i++) {
        y_i = y[i];
        y_p = getPrediction(x_i, w);
        if(y_i > 0 && y_p > 0)
            tp++;
        else if(y_i > 0 && y_p < 0)
            fn++;
        else if(y_i < 0 && y_p > 0)
            fp++;
        else
            tn++;
        x_i = (x_i + FEATURE_COUNT);
    }

    p = 0;
    r = 0;
    f1 = 0;
    if(tp > 0) {
        p = (double)tp / (double)(tp + fp);
        r = (double)tp / (double)(tp + fn);
        f1 = 2 * p * r / (p + r);
    }
    else {
        if(fp == 0)
            p = 1;
        if(fn == 0)
            r = 1;
    }

    accuracy = (double)(tp + tn) / (double)count;
    printf("tp, fp, tn, fn: %d, %d, %d, %d\n", tp, fp, tn, fn);
    printf("accuracy: %f\nf1: %f\n", accuracy, f1);
}

/**
 * getPrediction
 */
static double getPrediction(
    const double * const x_i,
    const double * const w
) {
    int i;
    double dot = 0;
    for(i = 0; i < FEATURE_COUNT; i++)
        dot += x_i[i] * w[i];
    return dot > 0 ? 1 : -1;
}

/**
 * printUsage
 */
static void printUsage(const char *prgm) {
    printf("usage:\n");
    printf("\t%s [-e <int>] [-C <double>] [-s <double>] [-g <double>]\n\n", prgm);
    printf("Options:\n");
    printf("\t-e <int>       Specifies the number of epochs over which to"
        " train.\n");
    printf("\t-C <double>    Specifies the C hyper parameter.\n");
    printf("\t-g <double>    Specifies the gamma0 hyper parameter.\n");
    printf("\t-s <double>    Specifies the s hyper parameter.\n");
}

/**
 * parseArgs
 */
static int parseArgs(
    const int argc,
    char **argv,
    int *epochs,
    double *c,
    double *gamma0,
    double *s
) {
    int i;
    for(i = 1; i < argc; i++) {
        /*** epochs ***/
        if(strcmp(argv[i], "-e") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -1;
            }
            (*epochs) = atoi(argv[i]);
            if((*epochs) < 1) {
                fprintf(stderr, "error: number of epochs must be greater than"
                    " 0\n");
                printUsage(argv[0]);
                return -2;
            }
        }
        /*** C ***/
        else if(strcmp(argv[i], "-C") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -3;
            }
            (*c) = atof(argv[i]);
            if((*c) <= 0) {
                fprintf(stderr, "error: C must be positive\n");
                printUsage(argv[0]);
                return -4;
            }
        }
        /*** gamma0 ***/
        else if(strcmp(argv[i], "-g") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -5;
            }
            (*gamma0) = atof(argv[i]);
            if((*gamma0) <= 0) {
                fprintf(stderr, "error: gamma0 must be positive\n");
                printUsage(argv[0]);
                return -6;
            }
        }
        /*** s ***/
        else if(strcmp(argv[i], "-s") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -7;
            }
            (*s) = atof(argv[i]);
            if((*s) <= 0) {
                fprintf(stderr, "error: s must be positive\n");
                printUsage(argv[0]);
                return -8;
            }
        }
        /*** unexpected argument ***/
        else {
            fprintf(stderr, "error: unexpected argument\n");
            printUsage(argv[0]);
            return -9;
        }
    }
    return 0;
}
