#include "hip/hip_runtime.h"
/*******************************************************************************
File: seq.c
Created by: CJ Dimaano
Date created: March 29, 2017
*******************************************************************************/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

extern "C" {
#include "data.h"
#include "mem.h"
}

/** Declarations **************************************************************/

static int train(
    double * const x,
    double * const y,
    const int count,
    const int epochs,
    const double c,
    const double gamma0,
    const double s,
    double * const w
);
static void test(
    const double * const x,
    const double * const y,
    const int count,
    const double * const w
);
static double getPrediction(
    const double * const x_i,
    const double * const w
);
static int parseArgs(const int, char **, int *, double *, double *, double *);

/** Main **********************************************************************/

int main(int argc, char **argv) {
    double *w, *x, *y, c = 1 << 10, gamma0 = 0.01, s = 10e5;
    int ret, epochs = 200;

    /*** Parse command-line arguments. ***/
    ret = parseArgs(argc, argv, &epochs, &c, &gamma0, &s);
    if(ret < 0) {
        return -1;
    }
    printf("epochs: %d\n", epochs);
    printf("C: %f\n", c);
    printf("gamma0: %f\n", gamma0);
    printf("s: %f\n", s);

    /*** Allocate memory. ***/
    if(init(&x, &y, &w) < 0) {
        return -2;
    }

    /*** Load training data. ***/
    ret = load(TRAIN_SET, x, y);
    if(ret < 0) {
        cleanup(&x, &y, &w);
        return -3;
    }

    /*** Train classifier. ***/
    ret = train(x, y, ret, epochs, c, gamma0, s, w);
    if(ret < 0) {
        cleanup(&x, &y, &w);
        return -4;
    }

    /*** Load test data. ***/
    ret = load(TEST_SET, x, y);
    if(ret < 0) {
        cleanup(&x, &y, &w);
        return ret;
    }

    /*** Test classifier accuracy. ***/
    test(x, y, ret, w);

    /*** Cleanup memory from examples. ***/
    cleanup(&x, &y, &w);

    return 0;
}

/** Static functions **********************************************************/

extern __global__ void trainCompute(double *p_w, double *p_x, double gamma0, double a, double b, double c, double e, int i, double t) {
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int j = tx + 32 * bx;

  if (j < FEATURE_COUNT) {
    p_w[j] = p_w[j] - (gamma0 / (1 + gamma0 * t / c)) * (a * p_x[i * FEATURE_COUNT + j] + b * p_w[j]);
  }

  __syncthreads();
}

void generateMoves(int count, int * moves) {
  int i, j, tmp;

  for (i = 0; i < count; i++) {
    moves[i] = i;
  }

  srand(time(NULL));
  for (i = 0; i < count; i++) {
    j = rand() % count;
    tmp = moves[i];
    moves[i] = moves[j];
    moves[j] = tmp;
  }
}

extern __global__ void performMoves(int count, int * moves, double * p_x, double * p_x_new, double * p_y, double *p_y_new) {
  int tid = threadIdx.x + 32 * blockIdx.x;

  if (tid < count) {
    memcpy((p_x_new + moves[tid] * FEATURE_COUNT), (p_x + tid * FEATURE_COUNT), FEATURE_COUNT * sizeof(double));
    p_y_new[moves[tid]] = p_y[tid];
  }
  __syncthreads();
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

int compare(double *a, double *b, int size, double threshold) {
  int i;
  for (i=0; i<size; i++) {
    if (abs(a[i]-b[i]) > threshold) return 0;
  }
  return 1;
}


/**
 * train
 *
 * @summary
 *   Trains the weights of the classifier.
 */
static int train(
    double * const x,
    double * const y,
    const int count,
    const int epochs,
    const double c,
    const double gamma0,
    const double s,
    double * const w
) {
    int epoch, i, j;
    double a, b = 2 / (s * s), t = 1, dot, e;

    // Allocate buffers on the GPU.
    int x_size = MAX_EXAMPLES * FEATURE_COUNT * sizeof(double);
    int y_size = MAX_EXAMPLES * sizeof(double);
    int w_size = FEATURE_COUNT * sizeof(double);

    double * p_x;
    hipMalloc((void **)&p_x, x_size);

    double * p_x_new;
    hipMalloc((void **)&p_x_new, x_size);

    double * p_x_temp;

    int * moves = (int *)malloc(count * sizeof(int));

    int * p_moves;
    hipMalloc((void **)&p_moves, count * sizeof(int));

    double *p_y;
    hipMalloc((void **)&p_y, y_size);

    double *p_y_new;
    hipMalloc((void **)&p_y_new, y_size);

    double *p_y_temp;

    double * p_w;
    hipMalloc((void **)&p_w, w_size);

    // Copy host buffers into device buffers.
    hipMemcpy(p_x, x, x_size, hipMemcpyHostToDevice);
    hipMemcpy(p_w, w, w_size, hipMemcpyHostToDevice);

    // Perform the computation.
    dim3 shuffleGrid((count+31)/32, 1);
    dim3 shuffleBlock(32, 1);

    dim3 dimGrid((FEATURE_COUNT+31)/32, 1);
    dim3 dimBlock(32, 1);

    for(epoch = 0; epoch < epochs; epoch++) {

        generateMoves(count, moves);

        hipMemcpy(p_moves, moves, count * sizeof(int), hipMemcpyHostToDevice);
        performMoves<<<shuffleGrid,shuffleBlock>>>(count, p_moves, p_x, p_x_new, p_y, p_y_new);

        gpuErrchk( hipDeviceSynchronize() );

        hipMemcpy(y, p_y_new, y_size, hipMemcpyDeviceToHost);
        hipMemcpy(x, p_x_new, x_size, hipMemcpyDeviceToHost);

        gpuErrchk( hipDeviceSynchronize() );

        p_x_temp = p_x;
        p_x = p_x_new;
        p_x_new = p_x_temp;

        p_y_temp = p_y;
        p_y = p_y_new;
        p_y_new = p_y_temp;

        for(i = 0; i < count; i++) {
            dot = 0;
            for(j = 0; j < FEATURE_COUNT; j++)
                dot += x[i * FEATURE_COUNT + j] * w[j];
            e = exp(-y[i] * dot);
            a = -y[i] * e / (1 + e);

            trainCompute<<<dimGrid,dimBlock>>>(p_w, p_x, gamma0, a, b, c, e, i, t);
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );

            hipMemcpy(w, p_w, w_size, hipMemcpyDeviceToHost);

            t += 1.0;
        }
    }

    // Copy the result off of the GPU.
    hipMemcpy(w, p_w, w_size, hipMemcpyDeviceToHost);

    // Free the created buffers.
    hipFree(p_x);
    hipFree(p_x_new);
    hipFree(p_y);
    hipFree(moves);
    hipFree(p_w);

    return 0;
}

/**
 * test
 */
static void test(
    const double * const x,
    const double * const y,
    const int count,
    const double * const w
) {
    int i;
    const double *x_i = x;
    double y_i, y_p, p, r, f1, accuracy;
    /*** True/False Positive/Negative ***/
    int tp = 0;
    int fp = 0;
    int tn = 0;
    int fn = 0;

    for(i = 0; i < count; i++) {
        y_i = y[i];
        y_p = getPrediction(x_i, w);
        if(y_i > 0 && y_p > 0)
            tp++;
        else if(y_i > 0 && y_p < 0)
            fn++;
        else if(y_i < 0 && y_p > 0)
            fp++;
        else
            tn++;
        x_i = (x_i + FEATURE_COUNT);
    }

    p = 0;
    r = 0;
    f1 = 0;
    if(tp > 0) {
        p = (double)tp / (double)(tp + fp);
        r = (double)tp / (double)(tp + fn);
        f1 = 2 * p * r / (p + r);
    }
    else {
        if(fp == 0)
            p = 1;
        if(fn == 0)
            r = 1;
    }

    accuracy = (double)(tp + tn) / (double)count;
    printf("tp, fp, tn, fn: %d, %d, %d, %d\n", tp, fp, tn, fn);
    printf("accuracy: %f\nf1: %f\n", accuracy, f1);
}

/**
 * getPrediction
 */
static double getPrediction(
    const double * const x_i,
    const double * const w
) {
    int i;
    double dot = 0;
    for(i = 0; i < FEATURE_COUNT; i++)
        dot += x_i[i] * w[i];
    return dot > 0 ? 1 : -1;
}

/**
 * printUsage
 */
static void printUsage(const char *prgm) {
    printf("usage:\n");
    printf("\t%s [-e <int>] [-C <double>] [-s <double>] [-g <double>]\n\n", prgm);
    printf("Options:\n");
    printf("\t-e <int>       Specifies the number of epochs over which to"
        " train.\n");
    printf("\t-C <double>    Specifies the C hyper parameter.\n");
    printf("\t-g <double>    Specifies the gamma0 hyper parameter.\n");
    printf("\t-s <double>    Specifies the s hyper parameter.\n");
}

/**
 * parseArgs
 */
static int parseArgs(
    const int argc,
    char **argv,
    int *epochs,
    double *c,
    double *gamma0,
    double *s
) {
    int i;
    for(i = 1; i < argc; i++) {
        /*** epochs ***/
        if(strcmp(argv[i], "-e") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -1;
            }
            (*epochs) = atoi(argv[i]);
            if((*epochs) < 1) {
                fprintf(stderr, "error: number of epochs must be greater than"
                    " 0\n");
                printUsage(argv[0]);
                return -2;
            }
        }
        /*** C ***/
        else if(strcmp(argv[i], "-C") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -3;
            }
            (*c) = atof(argv[i]);
            if((*c) <= 0) {
                fprintf(stderr, "error: C must be positive\n");
                printUsage(argv[0]);
                return -4;
            }
        }
        /*** gamma0 ***/
        else if(strcmp(argv[i], "-g") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -5;
            }
            (*gamma0) = atof(argv[i]);
            if((*gamma0) <= 0) {
                fprintf(stderr, "error: gamma0 must be positive\n");
                printUsage(argv[0]);
                return -6;
            }
        }
        /*** s ***/
        else if(strcmp(argv[i], "-s") == 0) {
            i++;
            if(i == argc) {
                fprintf(stderr, "error: unexpected end of argument list\n");
                printUsage(argv[0]);
                return -7;
            }
            (*s) = atof(argv[i]);
            if((*s) <= 0) {
                fprintf(stderr, "error: s must be positive\n");
                printUsage(argv[0]);
                return -8;
            }
        }
        /*** unexpected argument ***/
        else {
            fprintf(stderr, "error: unexpected argument\n");
            printUsage(argv[0]);
            return -9;
        }
    }
    return 0;
}
